#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hipblas.h>
#include <hipsparse.h>
#include "properties.h"

extern double compute_time;

void* malloc_gpu(const size_t size){
	void* ptr;
	if(size > 0){
		hipMalloc((void**)&ptr,size);
	}
	assert(ptr != NULL);
	return ptr;
}

void free_gpu(void* ptr){
	assert(ptr != NULL);
	hipFree(ptr);
}

__global__ void mbv_coo(int* row,int* col,double* val,int nnz,double* x,double* y){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	while(tid < nnz){
		y[col[tid]] += val[tid]*x[row[tid]];
		__syncthreads();
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void mbv_csc(int* row,double* val,int* PB,int* PE,int ncol,double* x,double* y,char tag){
	int index = blockDim.x*blockIdx.x+threadIdx.x;
	if('l' == tag){
		if(index < ncol){
			int col_begin = PB[index];
			int col_end = PE[index];
			int i;
			double sum = 0.0;
			for (i=col_begin;i<col_end;i++)
			{
			  sum+=val[i]*x[row[i]];
			}
			y[index] = sum;
		}
	}else{
		if(index<ncol){
			int col_begin = PB[index];
			int col_end = PE[index];
			int i;
			for(i=col_begin;i<col_end;i++){
			  y[row[i]]+=val[i]*x[index];
			}
        }
	}
}

__global__ void mbv_dia(int* diag,double* val,int nrow,int ncol,int ndiag,int lda,double* x,double* y,char tag){
	int index = blockDim.x*blockIdx.x+threadIdx.x;
	if(tag == 'l'){
		if(index < ncol){
			double sum = 0.0;
			int j;
			for (j=0; j<ndiag; ++j){
				int offset = diag[j];
				int istart = offset >= 0?0:(-offset);
				int jstart = offset >= 0?offset:0;
				int N = 0; int v_offset = 0; int I_offset = 0;

				if((nrow-istart) > (ncol-jstart)){
					N = ncol - jstart;
					I_offset = istart - jstart;
					v_offset = j*lda - jstart;
				}else{
					N = nrow - istart;
					I_offset = istart - jstart;
					v_offset = j*lda + lda - jstart - N;
				}

				int jend = jstart + N;

				if ((index >= jstart) && (index < jend)){
					sum += val[index+v_offset] * x[index+I_offset];
				}
				if(index >= jend){
					continue;
				}
			}
			y[index] = sum;
		}
	}else{
		if(index < nrow){
          double sum = 0.0;
          int j;
          for (j=0; j<ndiag; ++j){
            int offset = diag[j];
            int istart = offset >= 0?0:(-offset);
            int jstart = offset >= 0?offset:0;
            int N = 0; int v_offset = 0; int J_offset = 0;

            if((nrow-istart) >= (ncol-jstart)){
              N = ncol - jstart;
              J_offset = jstart - istart;
              v_offset = j*lda - istart;
            }else{
              N = nrow - istart;
              J_offset = jstart - istart;
              v_offset = j*lda + lda - nrow;
            }

            int iend = istart + N;
            
            if ((index >= istart) && (index < iend)){
              sum += val[index+v_offset] * x[index+J_offset];
            }
            if(index >= iend){
              continue;
            }
          }
          y[index] = sum;
        }
	}
}

__global__ void mbv_bsc(int* row,double* val,int* PB,int* PE,int nrow,int ncol,int nb,int lda,double* x,double*y){
	int cols = blockDim.x*blockIdx.x+threadIdx.x;
	int nn = lda;
    if(cols < nb){
    	int col_begin = PB[cols];
		int col_end = PE[cols];
		int i;
		for(i=col_begin;i<col_end;i++)
		{
		  int p;
		  for (p=0;p<nn;p++){
		    int q;
		    for (q=0;q<nn;q++){
		        if((cols*nn+q < ncol) && (row[i]*nn+p < nrow)){
		          y[cols*nn+q]+=val[i*nn*nn+p*nn+q]*x[row[i]*nn+p];
		        }
		    }
		  }
		}
    }        
}

__global__ void mbv_bdi(int*bdiag,double* val,int nrow,int ncol,int nbdiag,int blda,int mb,int nb,int mm,int nn,double* x,double* y,char tag){
	int index = blockDim.x*blockIdx.x+threadIdx.x;
	if('l' == tag){
		if(index<nb) {
			int j;
			for (j=0; j<nbdiag; ++j){
			  int offset = bdiag[j];
			  int istart = offset >= 0?0:(-offset);
			  int jstart = offset >= 0?offset:0;
			  int N = 0; int v_offset = 0; int I_offset = 0;

			  if((mb-istart) > (nb-jstart)){
			    N = nb - jstart;
			    I_offset = istart - jstart;
			    v_offset = j*blda - jstart;
			  }else{
			    N = mb - istart;
			    I_offset = istart - jstart;
			    v_offset = j*blda + blda - jstart - N;
			  }

			  int jend = jstart + N;
			  
			  if ((index >= jstart) && (index < jend)){
			    int p;
			    for (p=0;p<nn;p++){
			      int q;
			      for (q=0;q<nn;q++){
			        if((index*nn+q < ncol) && ((index+I_offset)*mm+p < nrow)){
			          y[index*nn+q] += val[(index+v_offset)*mm*nn+nn*p+q] * x[(index+I_offset)*mm+p];
			        }
			      }
			    }
			  }
			  if(index >= jend){
			    continue;
			  }
			}
		}
	}else{
		if(index<mb) {
			int j;
			for (j=0; j<nbdiag; ++j){
			  int offset = bdiag[j];
			  int istart = offset >= 0?0:(-offset);
			  int jstart = offset >= 0?offset:0;
			  int N = 0; int v_offset = 0; int J_offset = 0;

			  if((mb-istart) >= (nb-jstart)){
			    N = nb - jstart;
			    J_offset = jstart - istart;
			    v_offset = j*blda - istart;
			  }else{
			    N = mb - istart;
			    J_offset = jstart - istart;
			    v_offset = j*blda + blda - mb;
			  }

			  int iend = istart + N;
			  
			  if ((index >= istart) && (index < iend)){
			    int p;
			    for (p=0;p<mm;p++){
			      int q;
			      for (q=0;q<nn;q++){
			        if((index*mm+p <nrow) && ((index+J_offset)*nn+q < ncol)){
			          y[index*mm+p] += val[(index+v_offset)*mm*nn+nn*p+q] * x[(index+J_offset)*nn+q];
			        }
			      }
			    }
			  }
			  if(index >= iend){
			    continue;
			  }
			}
		}
	}
}

__global__ void mbv_csr(int* row,double* val,int* PB,int* PE,int nrow,double* x,double* y){
	int index = blockDim.x*blockIdx.x+threadIdx.x;
	if(index < nrow){
		int row_begin = PB[index];
		int row_end = PE[index];
		int i;
		double sum = 0.0;
		for (i=row_begin;i<row_end;i++)
		{
		  y[index]+=val[i]*x[row[i]];
		}
	}
}

extern "C" void lmbv_coo(int* row,int* col,double* val,int nrow,int ncol,int nnz,double* x,double* y){
	int *dev_row,*dev_col;
	double *dev_val,*dev_x,*dev_y;

	dev_row = (int*)malloc_gpu(nnz*sizeof(int));
	dev_col = (int*)malloc_gpu(nnz*sizeof(int));
	dev_val = (double*)malloc_gpu(nnz*sizeof(double));
	dev_x = (double*)malloc_gpu(nrow*sizeof(double));
	dev_y = (double*)malloc_gpu(ncol*sizeof(double));

	hipMemcpy(dev_row,row,nnz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_col,col,nnz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,nnz*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,nrow*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,ncol*sizeof(double),hipMemcpyHostToDevice);

	mbv_coo<<<(nnz+4-1)/4,4>>>(dev_row,dev_col,dev_val,nnz,dev_x,dev_y);

	hipMemcpy(y,dev_y,ncol*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_row);
	free_gpu(dev_col);
	free_gpu(dev_val);
	free_gpu(dev_x);
	free_gpu(dev_y);
}

extern "C" void lmbv_csr(int* row,int* col,double* val,int nrow,int ncol,int nnz,double* x,double* y){
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseMatDescr_t mat_descr = 0;
	hipsparseStatus_t stat_t;
	clock_t begin,end;

	stat_t = hipsparseCreate(&cusparseHandle);
	stat_t = hipsparseCreateMatDescr(&mat_descr);
	stat_t = hipsparseSetMatIndexBase(mat_descr,HIPSPARSE_INDEX_BASE_ZERO);
	stat_t = hipsparseSetMatType(mat_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	if(HIPSPARSE_STATUS_SUCCESS != stat_t){
		printf("Error!\n");
	}

	int *dev_row_offset,*dev_col;
	double *dev_val,*dev_x,*dev_y;
	const double alpha = 1.0;
	const double beta = 0.0;

	dev_row_offset = (int*)malloc_gpu((nrow+1)*sizeof(int));
	dev_col = (int*)malloc_gpu(nnz*sizeof(int));
	dev_val = (double*)malloc_gpu(nnz*sizeof(double));
	dev_x = (double*)malloc_gpu(nrow*sizeof(double));
	dev_y = (double*)malloc_gpu(ncol*sizeof(double));

	hipMemcpy(dev_row_offset,row,(nrow+1)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_col,col,nnz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,nnz*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,nrow*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,ncol*sizeof(double),hipMemcpyHostToDevice);

	begin = clock();
	stat_t = hipsparseDcsrmv(cusparseHandle,HIPSPARSE_OPERATION_TRANSPOSE,nrow,ncol,nnz,&alpha,mat_descr,
							dev_val,dev_row_offset,dev_col,dev_x,&beta,dev_y);
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end-begin)/CLOCKS_PER_SEC;

	if(HIPSPARSE_STATUS_SUCCESS != stat_t){
		printf("Error!\n");
	}

	hipMemcpy(y,dev_y,ncol*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_row_offset);
	free_gpu(dev_col);
	free_gpu(dev_val);
	free_gpu(dev_x);
	free_gpu(dev_y);

	stat_t = hipsparseDestroyMatDescr(mat_descr);
	stat_t = hipsparseDestroy(cusparseHandle);
}

extern "C" void rmbv_csr(int* row,int* col,double* val,int nrow,int ncol,int nnz,double* x,double* y){
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseMatDescr_t mat_descr = 0;
	hipsparseStatus_t stat_t;
	clock_t begin,end;

	stat_t = hipsparseCreate(&cusparseHandle);
	stat_t = hipsparseCreateMatDescr(&mat_descr);
	stat_t = hipsparseSetMatIndexBase(mat_descr,HIPSPARSE_INDEX_BASE_ZERO);
	stat_t = hipsparseSetMatType(mat_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	if(HIPSPARSE_STATUS_SUCCESS != stat_t){
		printf("Error!\n");
	}

	int *dev_row_offset,*dev_col;
	double *dev_val,*dev_x,*dev_y;
	const double alpha = 1.0;
	const double beta = 0.0;

	dev_row_offset = (int*)malloc_gpu((nrow+1)*sizeof(int));
	dev_col = (int*)malloc_gpu(nnz*sizeof(int));
	dev_val = (double*)malloc_gpu(nnz*sizeof(double));
	dev_x = (double*)malloc_gpu(ncol*sizeof(double));
	dev_y = (double*)malloc_gpu(nrow*sizeof(double));

	hipMemcpy(dev_row_offset,row,(nrow+1)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_col,col,nnz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,nnz*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,ncol*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,nrow*sizeof(double),hipMemcpyHostToDevice);

	begin = clock();
	stat_t = hipsparseDcsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nrow,ncol,nnz,&alpha,mat_descr,
							dev_val,dev_row_offset,dev_col,dev_x,&beta,dev_y);
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;

	if(HIPSPARSE_STATUS_SUCCESS != stat_t){
		printf("Error!\n");
	}

	hipMemcpy(y,dev_y,nrow*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_row_offset);
	free_gpu(dev_col);
	free_gpu(dev_val);
	free_gpu(dev_x);
	free_gpu(dev_y);

	stat_t = hipsparseDestroyMatDescr(mat_descr);
	stat_t = hipsparseDestroy(cusparseHandle);
}

// extern "C" void rmbv_csr(int* row,double* val,int* PB,int* PE,int nrow,int ncol,int nnz,double* x,double* y){
// 	int *dev_row,*dev_PB,*dev_PE;
// 	double *dev_val,*dev_x,*dev_y;
// 	clock_t begin,end;
// 	dim3 BlockSize(GPU_BLOCK_SIZE);
// 	dim3 GridSize((nrow+GPU_BLOCK_SIZE-1)/GPU_BLOCK_SIZE);

// 	dev_row = (int*)malloc_gpu(nnz*sizeof(int));
// 	dev_val = (double*)malloc_gpu(nnz*sizeof(double));
// 	dev_PB = (int*)malloc_gpu(nrow*sizeof(int));
// 	dev_PE = (int*)malloc_gpu(nrow*sizeof(int));
// 	dev_x = (double*)malloc_gpu(ncol*sizeof(double));
// 	dev_y = (double*)malloc_gpu(nrow*sizeof(double));

// 	hipMemcpy(dev_row,row,nnz*sizeof(int),hipMemcpyHostToDevice);
// 	hipMemcpy(dev_val,val,nnz*sizeof(double),hipMemcpyHostToDevice);
// 	hipMemcpy(dev_PB,PB,nrow*sizeof(int),hipMemcpyHostToDevice);
// 	hipMemcpy(dev_PE,PE,nrow*sizeof(int),hipMemcpyHostToDevice);
// 	hipMemcpy(dev_x,x,ncol*sizeof(double),hipMemcpyHostToDevice);
// 	hipMemcpy(dev_y,y,nrow*sizeof(double),hipMemcpyHostToDevice);

// 	begin = clock();
// 	mbv_csr<<<GridSize,BlockSize>>>(dev_row,dev_val,dev_PB,dev_PE,nrow,dev_x,dev_y);
// 	hipDeviceSynchronize();
// 	end = clock();
// 	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;
	
// 	hipMemcpy(y,dev_y,nrow*sizeof(double),hipMemcpyDeviceToHost);

// 	free_gpu(dev_row);
// 	free_gpu(dev_val);
// 	free_gpu(dev_PB);
// 	free_gpu(dev_PE);
// 	free_gpu(dev_x);
// 	free_gpu(dev_y);
// }

extern "C" void lmbv_csc(int* row,double* val,int* PB,int* PE,int nrow,int ncol,int nnz,double* x,double* y){
	int *dev_row,*dev_PB,*dev_PE;
	double *dev_val,*dev_x,*dev_y;
	clock_t begin,end;
	dim3 BlockSize(GPU_BLOCK_SIZE);
	dim3 GridSize((ncol+GPU_BLOCK_SIZE-1)/GPU_BLOCK_SIZE);

	dev_row = (int*)malloc_gpu(nnz*sizeof(int));
	dev_val = (double*)malloc_gpu(nnz*sizeof(double));
	dev_PB = (int*)malloc_gpu(ncol*sizeof(int));
	dev_PE = (int*)malloc_gpu(ncol*sizeof(int));
	dev_x = (double*)malloc_gpu(nrow*sizeof(double));
	dev_y = (double*)malloc_gpu(ncol*sizeof(double));

	hipMemcpy(dev_row,row,nnz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,nnz*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_PB,PB,ncol*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_PE,PE,ncol*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,nrow*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,ncol*sizeof(double),hipMemcpyHostToDevice);

	begin = clock();
	mbv_csc<<<GridSize,BlockSize>>>(dev_row,dev_val,dev_PB,dev_PE,ncol,dev_x,dev_y,'l');
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;
	
	hipMemcpy(y,dev_y,ncol*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_row);
	free_gpu(dev_val);
	free_gpu(dev_PB);
	free_gpu(dev_PE);
	free_gpu(dev_x);
	free_gpu(dev_y);
}

extern "C" void rmbv_csc(int* row,double* val,int* PB,int* PE,int nrow,int ncol,int nnz,double* x,double* y){
	int *dev_row,*dev_PB,*dev_PE;
	double *dev_val,*dev_x,*dev_y;
	clock_t begin,end;
	dim3 BlockSize(GPU_BLOCK_SIZE);
	dim3 GridSize((ncol+GPU_BLOCK_SIZE-1)/GPU_BLOCK_SIZE);

	dev_row = (int*)malloc_gpu(nnz*sizeof(int));
	dev_val = (double*)malloc_gpu(nnz*sizeof(double));
	dev_PB = (int*)malloc_gpu(ncol*sizeof(int));
	dev_PE = (int*)malloc_gpu(ncol*sizeof(int));
	dev_x = (double*)malloc_gpu(ncol*sizeof(double));
	dev_y = (double*)malloc_gpu(nrow*sizeof(double));

	hipMemcpy(dev_row,row,nnz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,nnz*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_PB,PB,ncol*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_PE,PE,ncol*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,ncol*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,nrow*sizeof(double),hipMemcpyHostToDevice);

	begin = clock();
	mbv_csc<<<GridSize,BlockSize>>>(dev_row,dev_val,dev_PB,dev_PE,ncol,dev_x,dev_y,'r');
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;
	
	hipMemcpy(y,dev_y,nrow*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_row);
	free_gpu(dev_val);
	free_gpu(dev_PB);
	free_gpu(dev_PE);
	free_gpu(dev_x);
	free_gpu(dev_y);
}

extern "C" void lmbv_dia(int* diag,double* val,int nrow,int ncol,int ndiag,int lda,double* x,double* y){
	int *dev_diag;
	double *dev_val,*dev_x,*dev_y;
	clock_t begin,end;
	dim3 BlockSize(GPU_BLOCK_SIZE);
	dim3 GridSize((ncol+GPU_BLOCK_SIZE-1)/GPU_BLOCK_SIZE);

	dev_diag = (int*)malloc_gpu(ndiag*sizeof(int));
	dev_val = (double*)malloc_gpu(ndiag*lda*sizeof(double));
	dev_x = (double*)malloc_gpu(nrow*sizeof(double));
	dev_y = (double*)malloc_gpu(ncol*sizeof(double));

	hipMemcpy(dev_diag,diag,ndiag*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,ndiag*lda*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,nrow*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,ncol*sizeof(double),hipMemcpyHostToDevice);
	
	begin = clock();
	mbv_dia<<<GridSize,BlockSize>>>(dev_diag,dev_val,nrow,ncol,ndiag,lda,dev_x,dev_y,'l');
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;
	
	hipMemcpy(y,dev_y,ncol*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_diag);
	free_gpu(dev_val);
	free_gpu(dev_x);
	free_gpu(dev_y);
}

extern "C" void rmbv_dia(int* diag,double* val,int nrow,int ncol,int ndiag,int lda,double* x,double* y){
	int *dev_diag;
	double *dev_val,*dev_x,*dev_y;
	dim3 BlockSize(GPU_BLOCK_SIZE);
	dim3 GridSize((nrow+GPU_BLOCK_SIZE-1)/GPU_BLOCK_SIZE);

	dev_diag = (int*)malloc_gpu(ndiag*sizeof(int));
	dev_val = (double*)malloc_gpu(ndiag*lda*sizeof(double));
	dev_x = (double*)malloc_gpu(ncol*sizeof(double));
	dev_y = (double*)malloc_gpu(nrow*sizeof(double));

	hipMemcpy(dev_diag,diag,ndiag*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,ndiag*lda*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,ncol*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,nrow*sizeof(double),hipMemcpyHostToDevice);


	mbv_dia<<<GridSize,BlockSize>>>(dev_diag,dev_val,nrow,ncol,ndiag,lda,dev_x,dev_y,'r');
	
	hipMemcpy(y,dev_y,nrow*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_diag);
	free_gpu(dev_val);
	free_gpu(dev_x);
	free_gpu(dev_y);
}

extern "C" void lmbv_bsc(int* row,double* val,int* PB,int* PE,int mb,int nb,int nrow,int ncol,int bnnz,int lda,double* x,double* y){
	int *dev_row,*dev_PB,*dev_PE;
	double *dev_val,*dev_x,*dev_y;
	int dummy = lda*lda;
	clock_t begin,end;
	dim3 BlockSize(GPU_BLOCK_SIZE);
	dim3 GridSize((nb+GPU_BLOCK_SIZE-1)/GPU_BLOCK_SIZE);

	dev_row = (int*)malloc_gpu(bnnz*sizeof(int));
	dev_val = (double*)malloc_gpu(dummy*bnnz*sizeof(double));
	dev_PB = (int*)malloc_gpu(nb*sizeof(int));
	dev_PE = (int*)malloc_gpu(nb*sizeof(int));
	dev_x = (double*)malloc_gpu(nrow*sizeof(double));
	dev_y = (double*)malloc_gpu(ncol*sizeof(double));

	hipMemcpy(dev_row,row,bnnz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,bnnz*dummy*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_PB,PB,nb*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_PE,PE,nb*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,nrow*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,ncol*sizeof(double),hipMemcpyHostToDevice);

	begin = clock();
	mbv_bsc<<<GridSize,BlockSize>>>(dev_row,dev_val,dev_PB,dev_PE,nrow,ncol,nb,lda,dev_x,dev_y);
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;
	
	hipMemcpy(y,dev_y,ncol*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_row);
	free_gpu(dev_val);
	free_gpu(dev_PB);
	free_gpu(dev_PE);
	free_gpu(dev_x);
	free_gpu(dev_y);
}

extern "C" void rmbv_bsc(int* row,double* val,int* PB,int* PE,int mb,int nb,int nrow,int ncol,int bnnz,int lda,double* x,double* y){
	int *dev_row,*dev_PB,*dev_PE;
	double *dev_val,*dev_x,*dev_y;
	int dummy = lda*lda;
	clock_t begin,end;
	dim3 BlockSize(GPU_BLOCK_SIZE);
	dim3 GridSize((nb+GPU_BLOCK_SIZE-1)/GPU_BLOCK_SIZE);

	dev_row = (int*)malloc_gpu(bnnz*sizeof(int));
	dev_val = (double*)malloc_gpu(dummy*bnnz*sizeof(double));
	dev_PB = (int*)malloc_gpu(nb*sizeof(int));
	dev_PE = (int*)malloc_gpu(nb*sizeof(int));
	dev_x = (double*)malloc_gpu(nrow*sizeof(double));
	dev_y = (double*)malloc_gpu(ncol*sizeof(double));

	hipMemcpy(dev_row,row,bnnz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,bnnz*dummy*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_PB,PB,nb*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_PE,PE,nb*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,nrow*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,ncol*sizeof(double),hipMemcpyHostToDevice);

	begin = clock();
	mbv_bsc<<<GridSize,BlockSize>>>(dev_row,dev_val,dev_PB,dev_PE,nrow,ncol,nb,lda,dev_x,dev_y);
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;
	
	hipMemcpy(y,dev_y,ncol*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_row);
	free_gpu(dev_val);
	free_gpu(dev_PB);
	free_gpu(dev_PE);
	free_gpu(dev_x);
	free_gpu(dev_y);
}

extern "C" void rmbv_bsr(int* row,int* col,double* val,int mb,int nb,int nrow,int ncol,int bnnz,int lda,double* x,double* y){
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseMatDescr_t mat_descr = 0;
	hipsparseStatus_t stat_t;
	clock_t begin,end;

	stat_t = hipsparseCreate(&cusparseHandle);
	stat_t = hipsparseCreateMatDescr(&mat_descr);
	stat_t = hipsparseSetMatIndexBase(mat_descr,HIPSPARSE_INDEX_BASE_ZERO);
	stat_t = hipsparseSetMatType(mat_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	if(HIPSPARSE_STATUS_SUCCESS != stat_t){
		printf("Error1!\n");
	}

	int *dev_row_offset,*dev_col;
	double *dev_val,*dev_x,*dev_y;
	const double alpha = 1.0;
	const double beta = 0.0;
	int dummy = lda*lda;

	dev_row_offset = (int*)malloc_gpu((mb+1)*sizeof(int));
	dev_col = (int*)malloc_gpu(bnnz*sizeof(int));
	dev_val = (double*)malloc_gpu(bnnz*dummy*sizeof(double));
	dev_x = (double*)malloc_gpu(nb*lda*sizeof(double));
	dev_y = (double*)malloc_gpu(mb*lda*sizeof(double));

	hipMemcpy(dev_row_offset,row,(mb+1)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_col,col,bnnz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,bnnz*dummy*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,ncol*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,nrow*sizeof(double),hipMemcpyHostToDevice);

	begin = clock();	
	stat_t = hipsparseDbsrmv(cusparseHandle,HIPSPARSE_DIRECTION_ROW,HIPSPARSE_OPERATION_NON_TRANSPOSE,mb,nb,bnnz,&alpha,mat_descr,
							dev_val,dev_row_offset,dev_col,lda,dev_x,&beta,dev_y);
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;

	if(HIPSPARSE_STATUS_SUCCESS != stat_t){
		printf("Error!\n");
	}

	hipMemcpy(y,dev_y,nrow*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_row_offset);
	free_gpu(dev_col);
	free_gpu(dev_val);
	free_gpu(dev_x);
	free_gpu(dev_y);

	stat_t = hipsparseDestroyMatDescr(mat_descr);
	stat_t = hipsparseDestroy(cusparseHandle);
}

extern "C" void lmbv_bdi(int*bdiag,double* val,int nrow,int ncol,int nbdiag,int blda,int mb,int nb,int mm,int nn,double* x,double* y){
	int *dev_bdiag;
	double *dev_val,*dev_x,*dev_y;
	clock_t begin,end;
	dim3 BlockSize(GPU_BLOCK_SIZE);
	dim3 GridSize((nb+GPU_BLOCK_SIZE-1)/GPU_BLOCK_SIZE);

	dev_bdiag = (int*)malloc_gpu(nbdiag*sizeof(int));
	dev_val = (double*)malloc_gpu(nbdiag*blda*mm*nn*sizeof(double));
	dev_x = (double*)malloc_gpu(nrow*sizeof(double));
	dev_y = (double*)malloc_gpu(ncol*sizeof(double));

	hipMemcpy(dev_bdiag,bdiag,nbdiag*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,nbdiag*blda*mm*nn*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,nrow*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,ncol*sizeof(double),hipMemcpyHostToDevice);
	
	begin = clock();
	mbv_bdi<<<GridSize,BlockSize>>>(dev_bdiag,dev_val,nrow,ncol,nbdiag,blda,mb,nb,mm,nn,dev_x,dev_y,'l');
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;
	
	hipMemcpy(y,dev_y,ncol*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_bdiag);
	free_gpu(dev_val);
	free_gpu(dev_x);
	free_gpu(dev_y);
}

extern "C" void rmbv_bdi(int*bdiag,double* val,int nrow,int ncol,int nbdiag,int blda,int mb,int nb,int mm,int nn,double* x,double* y){
	int *dev_bdiag;
	double *dev_val,*dev_x,*dev_y;
	clock_t begin,end;
	dim3 BlockSize(GPU_BLOCK_SIZE);
	dim3 GridSize((mb+GPU_BLOCK_SIZE-1)/GPU_BLOCK_SIZE);

	dev_bdiag = (int*)malloc_gpu(nbdiag*sizeof(int));
	dev_val = (double*)malloc_gpu(nbdiag*blda*mm*nn*sizeof(double));
	dev_x = (double*)malloc_gpu(ncol*sizeof(double));
	dev_y = (double*)malloc_gpu(nrow*sizeof(double));

	hipMemcpy(dev_bdiag,bdiag,nbdiag*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_val,val,nbdiag*blda*mm*nn*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,ncol*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,nrow*sizeof(double),hipMemcpyHostToDevice);
	
	begin = clock();
	mbv_bdi<<<GridSize,BlockSize>>>(dev_bdiag,dev_val,nrow,ncol,nbdiag,blda,mb,nb,mm,nn,dev_x,dev_y,'r');
	hipDeviceSynchronize();
	end = clock();
	compute_time += (double)(end - begin)/CLOCKS_PER_SEC;
	
	hipMemcpy(y,dev_y,nrow*sizeof(double),hipMemcpyDeviceToHost);

	free_gpu(dev_bdiag);
	free_gpu(dev_val);
	free_gpu(dev_x);
	free_gpu(dev_y);
}